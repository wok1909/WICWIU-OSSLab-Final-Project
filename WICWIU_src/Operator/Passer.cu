
#include <hip/hip_runtime.h>
#ifdef __CUDNN__

#include "Passer.hpp"

template class Passer<float>;

__global__ void Passer_ForwardPropagate_kernel(int sizeOfResultImg, int sizeOfInputImg, int timesize, int batchsize, float *result, float *input, int preSize) {
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < sizeOfInputImg; idx += blockDim.x * gridDim.x) {
        for (int ba = 0; ba < batchsize; ba++) {
            result[ba * sizeOfResultImg + preSize + idx] = input[ba * sizeOfInputImg + idx];
        }
    }
}

template<typename DTYPE> int Passer<DTYPE>::ForwardPropagateOnGPU(int pTime) {
    int noBlock = 3, threadsPerBlock = 128;

    Tensor<DTYPE> *result = this->GetResult();
    Tensor<DTYPE> *input  = NULL;

    int timesize    = result->GetTimeSize();
    int batchsize   = result->GetBatchSize();
    int channelsize = result->GetChannelSize();
    int rowsize     = result->GetRowSize();
    int colsize     = result->GetColSize();

    Shape *resultTenShape = result->GetShape();

    int sizeOfPlane     = rowsize * colsize;
    int sizeOfResultImg = channelsize * sizeOfPlane;
    int sizeOfInputImg  = 0;

    DTYPE *result_gpu = result->GetGPUData();
    DTYPE *input_gpu  = NULL;

    int preSize          = 0;
    int inputChannelSize = 0;

    for (int opnum = 0; opnum < m_noOperator; opnum++) {
        input            = this->GetInput()[opnum]->GetResult();
        input_gpu        = input->GetGPUData();
        inputChannelSize = input->GetChannelSize();
        preSize          = opnum * sizeOfPlane;
        sizeOfInputImg   = inputChannelSize * sizeOfPlane;
        // std::cout << "check" << '\n';
        GetKernelParameters(sizeOfInputImg, &noBlock, &threadsPerBlock);
        // printf("%d, %d\n", noBlock, threadsPerBlock);

        Passer_ForwardPropagate_kernel << < noBlock, threadsPerBlock >> > (sizeOfResultImg, sizeOfInputImg, timesize, batchsize, result_gpu, input_gpu, preSize);
    }
    return TRUE;
}

__global__ void Passer_BackPropagate_kernel(int sizeOfResultImg, int sizeOfInputImg, int timesize, int batchsize, float *delta_gpu, float *input_delta_gpu, int preSize) {
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < sizeOfInputImg; idx += blockDim.x * gridDim.x) {
        for (int ba = 0; ba < batchsize; ba++) {
            input_delta_gpu[ba * sizeOfInputImg + idx] += delta_gpu[ba * sizeOfResultImg + idx + preSize];
        }
    }
}

template<typename DTYPE> int Passer<DTYPE>::BackPropagateOnGPU(int pTime) {
    int noBlock = 3, threadsPerBlock = 128;

    Tensor<DTYPE> *this_delta  = this->GetDelta();
    Tensor<DTYPE> *input_delta = NULL;

    int timesize    = this_delta->GetTimeSize();
    int batchsize   = this_delta->GetBatchSize();
    int channelsize = this_delta->GetChannelSize();
    int rowsize     = this_delta->GetRowSize();
    int colsize     = this_delta->GetColSize();

    Shape *resultTenShape = this_delta->GetShape();

    int sizeOfPlane     = rowsize * colsize;
    int sizeOfResultImg = channelsize * sizeOfPlane;
    int sizeOfInputImg  = 0;

    DTYPE *delta_gpu       = this_delta->GetGPUData();
    DTYPE *input_delta_gpu = NULL;

    int preSize          = 0;
    int inputChannelSize = 0;

    for (int opnum = 0; opnum < m_noOperator; opnum++) {
        input_delta      = this->GetInput()[opnum]->GetDelta();
        input_delta_gpu  = input_delta->GetGPUData();
        inputChannelSize = input_delta->GetChannelSize();
        preSize          = opnum * sizeOfPlane;;
        sizeOfInputImg   = inputChannelSize * sizeOfPlane;
        GetKernelParameters(sizeOfInputImg, &noBlock, &threadsPerBlock);


        Passer_BackPropagate_kernel << < noBlock, threadsPerBlock >> > (sizeOfResultImg, sizeOfInputImg, timesize, batchsize, delta_gpu, input_delta_gpu, preSize);
    }
    return TRUE;
}

#endif  // ifdef __CUDNN__
