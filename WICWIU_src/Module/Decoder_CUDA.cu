
#include <hip/hip_runtime.h>
#ifdef __CUDNN__

#include "LinearLayer.hpp"
#include "EmbeddingLayer.hpp"
#include "RecurrentLayer.hpp"
#include "LSTMLayer.hpp"
#include "GRULayer.hpp"
#include "Decoder.hpp"

template class Decoder<float>;

/*!
@class LRelu cuda
*/
__global__ void ForwardPropagate_kernel(float *pDevEncoderHidden, float *pDevinitHidden, int batchIndex, int colSize) {
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < colSize; idx += blockDim.x * gridDim.x) {

          int startIndex = colSize * batchIndex;

          pDevinitHidden[startIndex + idx] = pDevEncoderHidden[startIndex + idx];

    }
}
/*!
  EncoderLengths가 NULL일때는 고려하지 않고 구현!!!
  추후 추가 필요!
*/

template<typename DTYPE> int Decoder<DTYPE>::ForwardPropagateOnGPU(int pTime) {


      if(pTime == 0){

          int noBlock = 3, threadsPerBlock = 128;

          Tensor<DTYPE> *encoderLengths = m_EncoderLengths->GetResult();

          //Data 접근!
          Tensor<DTYPE> *_initHidden = this->GetInput()[1]->GetResult();
          Tensor<DTYPE> *initHidden = m_initHiddenTensorholder->GetResult();

          // std::cout<<"Encoder last hidden value"<<'\n';
          // std::cout<<_initHidden->GetShape()<<'\n';
          // std::cout<<_initHidden<<'\n';

          //batchsize, colsize
          int batchsize  = _initHidden->GetBatchSize();
          int colSize    = _initHidden->GetColSize();

          // std::cout<<"복사 전"<<'\n';
          // std::cout<<initHidden->GetShape()<<'\n';
          // std::cout<<initHidden<<'\n';

          // std::cout<<"batch size : "<<batchsize<<'\n';

          for(int ba = 0; ba < batchsize; ba++){

              DTYPE *m_pDevEncoderHidden  = _initHidden->GetGPUData((*encoderLengths)[ba]-1);
              DTYPE *m_pDevinitHidden  = initHidden->GetGPUData(0);

              ForwardPropagate_kernel << < noBlock, threadsPerBlock >> > (m_pDevEncoderHidden, m_pDevinitHidden, ba, colSize);

          }

          // std::cout<<"복사 해온 값"<<'\n';
          // std::cout<<initHidden->GetShape()<<'\n';
          // std::cout<<initHidden<<'\n';
          // //
          // std::cout<<"Encoder length"<<'\n';
          // std::cout<<encoderLengths<<'\n';

      }

      int numOfExcutableOperator = this->GetNumOfExcutableOperator();
      Container<Operator<DTYPE> *> *ExcutableOperator = this->GetExcutableOperatorContainer();

      for (int i = 0; i < numOfExcutableOperator; i++) {
          (*ExcutableOperator)[i]->ForwardPropagateOnGPU(pTime);
      }
      return TRUE;
}


__global__ void BackPropagate_kernel(float *pDevEncoderHidden, float *pDevDecoderHidden, int batchIndex, int colSize) {
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < colSize; idx += blockDim.x * gridDim.x) {

          int startIndex = colSize * batchIndex;

          pDevEncoderHidden[startIndex + idx] = pDevDecoderHidden[startIndex + idx];

    }
}


template<typename DTYPE> int Decoder<DTYPE>::BackPropagateOnGPU(int pTime) {

      //seq2seq에 맞춰서 수정하기!!! --> Decoder안에 embedding, rnn, linear 다 있어서 layer별로 time을 다 실행하고 나서 넘어가야 됨!
      if(pTime !=0)
        return TRUE;
      //그래서 딱 한번만 호출되고 안쪽에서 모든 time에 대해 처리하도록!

      int numOfExcutableOperator = this->GetNumOfExcutableOperator();
      Container<Operator<DTYPE> *> *ExcutableOperator = this->GetExcutableOperatorContainer();

      // std::cout<<"backward 호출 전 initHidden gradient 값"<<'\n';
      // std::cout<<m_initHiddenTensorholder->GetGradient()<<'\n';

      // for (int i = numOfExcutableOperator - 1; i >= 0; i--) {
      //     (*ExcutableOperator)[i]->BackPropagateOnGPU(pTime);                   // 여기 어차피 한번만 돌려버리면 되는거 아닌가  --> embedding도 있어서 안됨!
      // }

      //seq2seq에 맞춰서 수정하기!!! --> Decoder안에 embedding, rnn, linear 다 있어서 layer별로 time을 다 실행하고 나서 넘어가야 됨!
      for (int i = numOfExcutableOperator - 1; i >= 0; i--) {
          for(int ti = timesize-1; ti>=0; ti--){
            (*ExcutableOperator)[i]->BackPropagateOnGPU(ti);
        }
      }


      //decoder에서 encoder로 복사!
      if(pTime == 0){

          int noBlock = 3, threadsPerBlock = 128;

          Tensor<DTYPE> *encoderLengths = m_EncoderLengths->GetResult();

          //Data 접근!
          Tensor<DTYPE> *_initHidden = this->GetInput()[1]->GetGradient();
          Tensor<DTYPE> *initHidden = m_initHiddenTensorholder->GetGradient();

          //batchsize, colsize
          int batchsize  = _initHidden->GetBatchSize();
          int colSize    = _initHidden->GetColSize();

          // std::cout<<"복사 전 encoder의 gradient"<<'\n';
          // std::cout<<_initHidden<<'\n';

          // std::cout<<"initHidden의 gradient"<<'\n';
          // std::cout<<initHidden<<'\n';

          for(int ba = 0; ba < batchsize; ba++){

              DTYPE *m_pDevEncoderHidden  = _initHidden->GetGPUData((*encoderLengths)[ba]-1);
              DTYPE *m_pDevinitHidden  = initHidden->GetGPUData(0);

              BackPropagate_kernel << < noBlock, threadsPerBlock >> > (m_pDevEncoderHidden, m_pDevinitHidden, ba, colSize);

          }

          // std::cout<<"복사 후 encoder의 gradient"<<'\n';
          // std::cout<<_initHidden<<'\n';
          //
          //
          // std::cout<<"Encoder length"<<'\n';
          // std::cout<<encoderLengths<<'\n';

      }

      return TRUE;
}

#endif  // ifdef __CUDNN__
